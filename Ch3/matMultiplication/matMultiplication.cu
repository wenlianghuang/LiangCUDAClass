
#include <hip/hip_runtime.h>
#include<iostream>


void RandomInit(float *mat,int N)
{
   for(int i = 0; i < N*N; i++)
      mat[i] = rand()/(float) RAND_MAX;
}

__global__ void matMul(float *matA, float *matB, float *matC, int N)
{
   int column = threadIdx.x + blockIdx.x*blockDim.x;
   int row = threadIdx.y + blockIdx.x*blockDim.y;
   
   int sum = 0;
   if(column < N && row < N)
   {
      for(int k = 0; k < N*N; k++)
      {
         sum += matA[N*column + k] * matB[k*N + column];
      }
      
      matC[row*N+column] = sum;
   }
} 
int main()
{
   int matrixSize;
   printf("Input the matrix size: ");
   scanf("%d",&matrixSize);
   
   size_t size = matrixSize * matrixSize * sizeof(int);
   float *matA,*matB,*matC,*matD;
   float *gpuMatA,*gpuMatB,*gpuMatC;
  
   matA = (float*)malloc(size);
   matB = (float*)malloc(size);
   matC = (float*)malloc(size);

   RandomInit(matA,matrixSize);
   RandomInit(matB,matrixSize);

   hipEvent_t gpuStart,gpuStop,cpuStart,cpuStop;
   hipEventCreate(&gpuStart);
   hipEventCreate(&gpuStop);
   hipEventCreate(&cpuStart);
   hipEventCreate(&cpuStop);

   float cpu_tottime,gpu_tottime;
   hipEventRecord(cpuStart,0);
   
   for(int i = 0; i < matrixSize; i++)
   {
      for(int j = 0; j < matrixSize; j++)
      {
         int sum = 0;
         for(int k = 0; k < matrixSize; k++)
         {
            sum += matA[i*matrixSize + k] * matB[k*matrixSize + j];
         }
         matC[i*matrixSize + j] = sum;
      }
   }

   hipEventRecord(cpuStop,0);
   hipEventSynchronize(cpuStop);
   hipEventElapsedTime(&cpu_tottime,cpuStart,cpuStop);
   printf("CPU time %5.5f (ms) by matrix multiplication\n",cpu_tottime);

   int threadsPerblock = 0;
   int blocksPergrid = 0;
   hipEventRecord(gpuStart,0);
   hipMalloc((void**)&gpuMatA,size);
   hipMalloc((void**)&gpuMatB,size);
   hipMalloc((void**)&gpuMatC,size);

   hipMemcpy((void **)gpuMatA,matA,size,hipMemcpyHostToDevice);
   hipMemcpy((void **)gpuMatB,matB,size,hipMemcpyHostToDevice);

   hipEventRecord(gpuStop,0);
   hipEventSynchronize(gpuStop);

   float Inittime;
   hipEventElapsedTime(&Inittime,gpuStart,gpuStop);
   printf("Input time: %5.5f (ms)\n",Inittime);

   printf("Input the threads per block: ");
   scanf("%d",&threadsPerblock);
   printf("\nInput the blocks per grid: ");
   scanf("%d",&blocksPergrid);
   printf("\n");
   
   matD = (float*)malloc(size);
   dim3 blocks(threadsPerblock,threadsPerblock);
   dim3 grid(blocksPergrid,blocksPergrid);
   
   hipEventRecord(gpuStart,0);
   matMul<<<grid,blocks>>>(gpuMatA,gpuMatB,gpuMatC,matrixSize);
   hipEventRecord(gpuStop,0);
   hipEventSynchronize(gpuStop);

   float pro_time;
   hipEventElapsedTime(&pro_time,gpuStart,gpuStop);
   printf("GPU Processing time: %5.5f (ms)\n",pro_time);

   hipEventRecord(gpuStart,0); 
   hipMemcpy(matD,gpuMatC,size,hipMemcpyDeviceToHost);
   hipFree(gpuMatA);
   hipFree(gpuMatB);
   hipFree(gpuMatC);
   hipEventRecord(gpuStop,0);
   hipEventSynchronize(gpuStop);
   float Outtime;
   hipEventElapsedTime(&Outtime,gpuStart,gpuStop);
   printf("Output time: %5.5f (ms)\n",Outtime);

   printf("Total time of GPU: %5.5f (ms)\n",(Inittime+pro_time+Outtime));
   
}



   
    
