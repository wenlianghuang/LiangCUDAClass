// Vector Dot Product A.B 
// using multiple GPUs with OpenMP

// Includes
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>          // header for OpenMP
#include <hip/hip_runtime.h>

// Variables
float* h_A;   // host vectors
float* h_B;
float* h_C;
float h_D = 0.0;
float h_G = 0.0;
// Functions
void RandomInit(float*, int);

// Device code
__global__ void VecDot(const float* A, const float* B, float* C, int N)
{
    extern __shared__ float cache[];

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int cacheIdx = threadIdx.x;

    float temp = 0.0;
    while ( i < N)
    {
        temp += A[i] * B[i];
        i += blockDim.x * gridDim.x;
    }

    cache[cacheIdx] = temp;

    __syncthreads();

    int ib = blockDim.x/2;
    while(ib!=0)
    {
        if(cacheIdx < ib)
            cache[cacheIdx] += cache[cacheIdx + ib];
        __syncthreads();

        ib /= 2;
    }

    if(cacheIdx ==0)
        C[blockIdx.x] = cache[0];
}



// Host code

int main(void)
{
    printf("\n");
    printf("Vector Dot with multiple GPUs \n");
    int N, NGPU, cpu_thread_id=0;
    int *Dev; 
    long mem = 1024*1024*1024;     // 4 Giga for float data type.

    printf("Enter the number of GPUs: ");
    scanf("%d", &NGPU);
    printf("%d\n", NGPU);
    Dev = (int *)malloc(sizeof(int)*NGPU); //Make sure to the number of GPU

    int numDev = 0;
    printf("GPU device number: ");
    for(int i = 0; i < NGPU; i++) {
      scanf("%d", &Dev[i]);
      printf("%d ",Dev[i]);
      numDev++;
      if(getchar() == '\n') break;
    }
    printf("\n");
    if(numDev != NGPU) {
      fprintf(stderr,"Should input %d GPU device numbers\n", NGPU);
      exit(1);
    }

    printf("Enter the size of the vectors: ");
    scanf("%d", &N);        
    printf("%d\n", N);        
    
    /**
    if (3*N > mem) {
        printf("The size of these 3 vectors cannot be fitted into 4 Gbyte\n");
        exit(1);
    }**/

    int size = N*sizeof(float);
    
    // Set the sizes of threads and blocks
    int threadsPerBlock;
    printf("Enter the number of threads per block: ");
    scanf("%d", &threadsPerBlock);
    printf("%d\n", threadsPerBlock);
    if(threadsPerBlock > 1024) {
      printf("The number of threads per block must be less than 1024 ! \n");
      exit(1);
    }
    /**int blocksPerGrid = (N + threadsPerBlock*NGPU - 1) / (threadsPerBlock*NGPU);**/
    int blocksPerGrid;
    printf("Eneter the number of the blocks per grid: ");
    scanf("%d",&blocksPerGrid);
    printf("%d\n", blocksPerGrid);
    if(blocksPerGrid > 2147483647) {
      printf("The number of blocks must be less than 2147483647 ! \n");
      exit(1);
    }
    int sb = blocksPerGrid * sizeof(float);
    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    if (! h_A || ! h_B || ! h_C) {
        printf("!!! Not enough memory.\n");
        exit(1);
    }
    
    // Initialize input vectors

    RandomInit(h_A, N);
    RandomInit(h_B, N);

    // declare cuda event for timer
    hipEvent_t start, stop;
//    cudaEventCreate(&start);    // events must be created after devices are set 
//    cudaEventCreate(&stop);

    float Intime,gputime,Outime;

    omp_set_num_threads(NGPU);

    #pragma omp parallel private(cpu_thread_id)
    {
	float *d_A, *d_B, *d_C;
	cpu_thread_id = omp_get_thread_num();
	printf("cpu_thread_id: %d\n",cpu_thread_id);
    hipSetDevice(Dev[cpu_thread_id]);
    // cudaSetDevice(cpu_thread_id);
    // start the timer
    if(cpu_thread_id == 0) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start,0);
    }

	// Allocate vectors in device memory
	hipMalloc((void**)&d_A, size/NGPU);
	hipMalloc((void**)&d_B, size/NGPU);
	hipMalloc((void**)&d_C, size/NGPU);
    
    printf("sb put into h_C\n");
    // Copy vectors from host memory to device memory
	hipMemcpy(d_A, h_A+N/NGPU*cpu_thread_id, size/NGPU, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B+N/NGPU*cpu_thread_id, size/NGPU, hipMemcpyHostToDevice);
	printf("hipMemcpy finish\n");
    #pragma omp barrier

    // stop the timer
	if(cpu_thread_id == 0) {
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &Intime, start, stop);
        printf("Data input time for GPU: %f (ms) \n",Intime);
    }

    // start the timer
    if(cpu_thread_id == 0) hipEventRecord(start,0);
    int sm = threadsPerBlock*sizeof(float);
    VecDot<<<blocksPerGrid, threadsPerBlock,sm>>>(d_A, d_B, d_C, N/NGPU);
	hipDeviceSynchronize();
    
    // stop the timer
	if(cpu_thread_id == 0) {
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &gputime, start, stop);
        printf("Processing time for GPU: %f (ms) \n",gputime);
        printf("GPU Gflops: %f\n",(2*N-1)/(1000000.0*gputime));
    }
    
    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    
    // start the timer
    if(cpu_thread_id == 0) hipEventRecord(start,0);

    /**cudaMemcpy(h_C+N/NGPU*cpu_thread_id, d_C, size/NGPU, cudaMemcpyDeviceToHost);**/
    hipMemcpy(h_C+N/NGPU*cpu_thread_id, d_C, size/NGPU, hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

    //float h_G = 0.0;
    for(int i =0; i < blocksPerGrid; i++)
    {
        h_G += h_C[i];
    }
        
    // stop the timer
	if(cpu_thread_id == 0) {
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &Outime, start, stop);
        printf("Data output time for GPU: %f (ms) \n",Outime);
    }
    printf("Test0\n");
    } 

    float gputime_tot;
    gputime_tot = Intime + gputime + Outime;
    printf("Test1\n");
    printf("Total time for GPU: %f (ms) \n",gputime_tot);

    // start the timer
    hipEventRecord(start,0);

    //h_D = (float*)malloc(size);     // compute the reference solution
    
    for (int i = 0; i < N; ++i) 
        h_D += (double) h_A[i] * h_B[i];
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float cputime;
    hipEventElapsedTime( &cputime, start, stop);
    printf("Processing time for CPU: %f (ms) \n",cputime);
    printf("CPU Gflops: %f\n",(2*N-1)/(1000000.0*cputime));
    printf("Speed up of GPU = %f\n", cputime/gputime_tot);

    // Destroy timer
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // check result
    printf("Check result:\n");
    double sum=0; 
    double diff = abs((h_G-h_D)/h_G);
    /**
    for (int i = 0; i < N; ++i) {
        diff = abs(h_D[i] - h_G[i]);
        sum += diff*diff; 
    }
    sum = sqrt(sum);
    **/
    
    printf("|h_G - h_D|/h_G=%20.15e\n",diff);
    printf("h_D = %20.15e\n",h_D);
    printf("h_G = %20.15e\n",h_G);
    for (int i=0; i < NGPU; i++) {
	    hipSetDevice(i);
	    hipDeviceReset();
    }

    return 0;
}


// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = 2.0*rand() / (float)RAND_MAX - 1.0;
}
