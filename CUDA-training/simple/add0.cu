//: nvcc add0.cu -o add0


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void cuda_add(int a, int b, int *c)
{
    *c = a + b;
}

int main(int argc, char **argv)
{
    int c;
    int *dev_c;
    hipMalloc((void**)&dev_c, sizeof(int));
    cuda_add<<<1,1>>>(2, 2, dev_c);
    /*
     * Arguments pour cudaMemcpy
     * 1 : destination
     * 2 : memoire sur le device
     * 3 : taille du bloc
     * 4 : direction de la copie
     */
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("Almighty CUDA's answer: 2 + 2 = %d.\n", c);
    hipFree(dev_c);
    return EXIT_SUCCESS;
}

