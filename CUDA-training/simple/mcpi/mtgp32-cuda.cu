#include "hip/hip_runtime.h"
/*
 * @file mtgp32-cuda.cu
 *
 * @brief Sample Program for CUDA 3.2 and 4.0
 *
 * MTGP32-11213
 * This program generates 32-bit unsigned integers.
 * The period of generated integers is 2<sup>11213</sup>-1.
 *
 * This also generates single precision floating point numbers
 * uniformly distributed in the range [1, 2). (float r; 1.0 <= r < 2.0)
 */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <inttypes.h>
#include <errno.h>
#include <stdlib.h>

#include "mtgp-util.cuh"
#include "mtgp32-fast.h"

#define MTGPDC_MEXP 11213
#define MTGPDC_N 351
#define MTGPDC_FLOOR_2P 256
#define MTGPDC_CEIL_2P 512
#define MTGPDC_PARAM_TABLE mtgp32dc_params_fast_11213
#define MEXP 11213
#define THREAD_NUM MTGPDC_FLOOR_2P
#define LARGE_SIZE (THREAD_NUM * 3)
#define BLOCK_NUM_MAX 200
#define TBL_SIZE 16
#define N MTGPDC_N

extern mtgp32_params_fast_t mtgp32dc_params_fast_11213[];

/**
 * kernel I/O
 * This structure must be initialized before first use.
 */
struct mtgp32_kernel_status_t {
    uint32_t status[MTGPDC_N];
};

/*
 * Generator Parameters.
 */
__constant__ unsigned int pos_tbl[BLOCK_NUM_MAX];
__constant__ uint32_t param_tbl[BLOCK_NUM_MAX][TBL_SIZE];
__constant__ uint32_t temper_tbl[BLOCK_NUM_MAX][TBL_SIZE];
__constant__ uint32_t single_temper_tbl[BLOCK_NUM_MAX][TBL_SIZE];
__constant__ uint32_t sh1_tbl[BLOCK_NUM_MAX];
__constant__ uint32_t sh2_tbl[BLOCK_NUM_MAX];
__constant__ uint32_t mask[1];

/**
 * Shared memory
 * The generator's internal status vector.
 */
__shared__ uint32_t status[LARGE_SIZE];

/**
 * The function of the recursion formula calculation.
 *
 * @param[in] X1 the farthest part of state array.
 * @param[in] X2 the second farthest part of state array.
 * @param[in] Y a part of state array.
 * @param[in] bid block id.
 * @return output
 */
__device__ uint32_t para_rec(uint32_t X1, uint32_t X2, uint32_t Y, int bid) {
    uint32_t X = (X1 & mask[0]) ^ X2;
    uint32_t MAT;

    X ^= X << sh1_tbl[bid];
    Y = X ^ (Y >> sh2_tbl[bid]);
    MAT = param_tbl[bid][Y & 0x0f];
    return Y ^ MAT;
}

/**
 * The tempering function.
 *
 * @param[in] V the output value should be tempered.
 * @param[in] T the tempering helper value.
 * @param[in] bid block id.
 * @return the tempered value.
 */
__device__ uint32_t temper(uint32_t V, uint32_t T, int bid) {
    uint32_t MAT;

    T ^= T >> 16;
    T ^= T >> 8;
    MAT = temper_tbl[bid][T & 0x0f];
    return V ^ MAT;
}

/**
 * The tempering and converting function.
 * By using the preset-ted table, converting to IEEE format
 * and tempering are done simultaneously.
 *
 * @param[in] V the output value should be tempered.
 * @param[in] T the tempering helper value.
 * @param[in] bid block id.
 * @return the tempered and converted value.
 */
__device__ uint32_t temper_single(uint32_t V, uint32_t T, int bid) {
    uint32_t MAT;
    uint32_t r;

    T ^= T >> 16;
    T ^= T >> 8;
    MAT = single_temper_tbl[bid][T & 0x0f];
    r = (V >> 9) ^ MAT;
    return r;
}

/**
 * Read the internal state vector from kernel I/O data, and
 * put them into shared memory.
 *
 * @param[out] status shared memory.
 * @param[in] d_status kernel I/O data
 * @param[in] bid block id
 * @param[in] tid thread id
 */
__device__ void status_read(uint32_t status[LARGE_SIZE],
			    const mtgp32_kernel_status_t *d_status,
			    int bid,
			    int tid) {
    status[LARGE_SIZE - N + tid] = d_status[bid].status[tid];
    if (tid < N - THREAD_NUM) {
	status[LARGE_SIZE - N + THREAD_NUM + tid]
	    = d_status[bid].status[THREAD_NUM + tid];
    }
    __syncthreads();
}

/**
 * Read the internal state vector from shared memory, and
 * write them into kernel I/O data.
 *
 * @param[out] d_status kernel I/O data
 * @param[in] status shared memory.
 * @param[in] bid block id
 * @param[in] tid thread id
 */
__device__ void status_write(mtgp32_kernel_status_t *d_status,
			     const uint32_t status[LARGE_SIZE],
			     int bid,
			     int tid) {
    d_status[bid].status[tid] = status[LARGE_SIZE - N + tid];
    if (tid < N - THREAD_NUM) {
	d_status[bid].status[THREAD_NUM + tid]
	    = status[4 * THREAD_NUM - N + tid];
    }
    __syncthreads();
}

/**
 * kernel function.
 * This function generates 32-bit unsigned integers in d_data
 *
 * @param[in,out] d_status kernel I/O data
 * @param[out] d_data output
 * @param[in] size number of output data requested.
 */
__global__ void mtgp32_uint32_kernel(mtgp32_kernel_status_t* d_status,
				     uint32_t* d_data, int size) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    int pos = pos_tbl[bid];
    uint32_t r;
    uint32_t o;

    // copy status data from global memory to shared memory.
    status_read(status, d_status, bid, tid);

    // main loop
    for (int i = 0; i < size; i += LARGE_SIZE) {

#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("status[LARGE_SIZE - N + tid]:%08x\n",
		   status[LARGE_SIZE - N + tid]);
	    printf("status[LARGE_SIZE - N + tid + 1]:%08x\n",
		   status[LARGE_SIZE - N + tid + 1]);
	    printf("status[LARGE_SIZE - N + tid + pos]:%08x\n",
		   status[LARGE_SIZE - N + tid + pos]);
	    printf("sh1:%d\n", sh1_tbl[bid]);
	    printf("sh2:%d\n", sh2_tbl[bid]);
	    printf("mask:%08x\n", mask[0]);
	    for (int j = 0; j < 16; j++) {
		printf("tbl[%d]:%08x\n", j, param_tbl[0][j]);
	    }
	}
#endif
	r = para_rec(status[LARGE_SIZE - N + tid],
		 status[LARGE_SIZE - N + tid + 1],
		 status[LARGE_SIZE - N + tid + pos],
		 bid);
	status[tid] = r;
#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("status[tid]:%08x\n", status[tid]);
	}
#endif
	o = temper(r, status[LARGE_SIZE - N + tid + pos - 1], bid);
#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("r:%08" PRIx32 "\n", r);
	}
#endif
	d_data[size * bid + i + tid] = o;
	__syncthreads();
	r = para_rec(status[(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		     status[(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		     status[(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		     bid);
	status[tid + THREAD_NUM] = r;
	o = temper(r,
		   status[(4 * THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
		   bid);
	d_data[size * bid + THREAD_NUM + i + tid] = o;
	__syncthreads();
	r = para_rec(status[2 * THREAD_NUM - N + tid],
		     status[2 * THREAD_NUM - N + tid + 1],
		     status[2 * THREAD_NUM - N + tid + pos],
		     bid);
	status[tid + 2 * THREAD_NUM] = r;
	o = temper(r, status[tid + pos - 1 + 2 * THREAD_NUM - N], bid);
	d_data[size * bid + 2 * THREAD_NUM + i + tid] = o;
	__syncthreads();
    }
    // write back status for next call
    status_write(d_status, status, bid, tid);
}

/**
 * kernel function.
 * This function generates single precision floating point numbers in d_data.
 *
 * @param[in,out] d_status kernel I/O data
 * @param[out] d_data output. IEEE single precision format.
 * @param[in] size number of output data requested.
 */
__global__ void mtgp32_single_kernel(mtgp32_kernel_status_t* d_status,
				     uint32_t* d_data, int size)
{

    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    int pos = pos_tbl[bid];
    uint32_t r;
    uint32_t o;

    // copy status data from global memory to shared memory.
    status_read(status, d_status, bid, tid);

    // main loop
    for (int i = 0; i < size; i += LARGE_SIZE) {
	r = para_rec(status[LARGE_SIZE - N + tid],
		     status[LARGE_SIZE - N + tid + 1],
		     status[LARGE_SIZE - N + tid + pos],
		     bid);
	status[tid] = r;
	o = temper_single(r, status[LARGE_SIZE - N + tid + pos - 1], bid);
	d_data[size * bid + i + tid] = o;
	__syncthreads();
	r = para_rec(status[(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		     status[(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		     status[(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		     bid);
	status[tid + THREAD_NUM] = r;
	o = temper_single(
	    r,
	    status[(4 * THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
	    bid);
	d_data[size * bid + THREAD_NUM + i + tid] = o;
	__syncthreads();
	r = para_rec(status[2 * THREAD_NUM - N + tid],
		     status[2 * THREAD_NUM - N + tid + 1],
		     status[2 * THREAD_NUM - N + tid + pos],
		     bid);
	status[tid + 2 * THREAD_NUM] = r;
	o = temper_single(r,
			  status[tid + pos - 1 + 2 * THREAD_NUM - N],
			  bid);
	d_data[size * bid + 2 * THREAD_NUM + i + tid] = o;
	__syncthreads();
    }
    // write back status for next call
    status_write(d_status, status, bid, tid);
}

/**
 * This function initializes kernel I/O data.
 * @param d_status output kernel I/O data.
 * @param params MTGP32 parameters. needed for the initialization.
 */
void make_kernel_data32(mtgp32_kernel_status_t * d_status,
			mtgp32_params_fast_t params[],
			int block_num)
{
    int i;
    mtgp32_kernel_status_t* h_status
	= (mtgp32_kernel_status_t *) malloc(
	    sizeof(mtgp32_kernel_status_t) * block_num);

    if (h_status == NULL) {
	printf("failure in allocating host memory for kernel I/O data.\n");
	exit(8);
    }
    for (i = 0; i < block_num; i++) {
	mtgp32_init_state(&(h_status[i].status[0]), &params[i], i + 1);
    }
#if defined(DEBUG)
    printf("h_status[0].status[0]:%08"PRIx32"\n", h_status[0].status[0]);
    printf("h_status[0].status[1]:%08"PRIx32"\n", h_status[0].status[1]);
    printf("h_status[0].status[2]:%08"PRIx32"\n", h_status[0].status[2]);
    printf("h_status[0].status[3]:%08"PRIx32"\n", h_status[0].status[3]);
#endif
    ccudaMemcpy(d_status, h_status,
		sizeof(mtgp32_kernel_status_t) * block_num,
		hipMemcpyHostToDevice);
    free(h_status);
}

/**
 * This function sets constants in device memory.
 * @param[in] params input, MTGP32 parameters.
 */
void make_constant(const mtgp32_params_fast_t params[],
    int block_num) {
    const int size1 = sizeof(uint32_t) * block_num;
    const int size2 = sizeof(uint32_t) * block_num * TBL_SIZE;
    uint32_t *h_pos_tbl;
    uint32_t *h_sh1_tbl;
    uint32_t *h_sh2_tbl;
    uint32_t *h_param_tbl;
    uint32_t *h_temper_tbl;
    uint32_t *h_single_temper_tbl;
    uint32_t *h_mask;
    h_pos_tbl = (uint32_t *)malloc(size1);
    h_sh1_tbl = (uint32_t *)malloc(size1);
    h_sh2_tbl = (uint32_t *)malloc(size1);
    h_param_tbl = (uint32_t *)malloc(size2);
    h_temper_tbl = (uint32_t *)malloc(size2);
    h_single_temper_tbl = (uint32_t *)malloc(size2);
    h_mask = (uint32_t *)malloc(sizeof(uint32_t));
    if (h_pos_tbl == NULL
	|| h_sh1_tbl == NULL
	|| h_sh2_tbl == NULL
	|| h_param_tbl == NULL
	|| h_temper_tbl == NULL
	|| h_single_temper_tbl == NULL
	|| h_mask == NULL
	) {
	printf("failure in allocating host memory for constant table.\n");
	exit(1);
    }
    h_mask[0] = params[0].mask;
    for (int i = 0; i < block_num; i++) {
	h_pos_tbl[i] = params[i].pos;
	h_sh1_tbl[i] = params[i].sh1;
	h_sh2_tbl[i] = params[i].sh2;
	for (int j = 0; j < TBL_SIZE; j++) {
	    h_param_tbl[i * TBL_SIZE + j] = params[i].tbl[j];
	    h_temper_tbl[i * TBL_SIZE + j] = params[i].tmp_tbl[j];
	    h_single_temper_tbl[i * TBL_SIZE + j] = params[i].flt_tmp_tbl[j];
	}
    }
    ccudaMemcpyToSymbol(pos_tbl, h_pos_tbl, size1);
    ccudaMemcpyToSymbol(sh1_tbl, h_sh1_tbl, size1);
    ccudaMemcpyToSymbol(sh2_tbl, h_sh2_tbl, size1);
    ccudaMemcpyToSymbol(param_tbl, h_param_tbl, size2);
    ccudaMemcpyToSymbol(temper_tbl, h_temper_tbl, size2);
    ccudaMemcpyToSymbol(single_temper_tbl, h_single_temper_tbl, size2);
    ccudaMemcpyToSymbol(mask, h_mask, sizeof(uint32_t));
    free(h_pos_tbl);
    free(h_sh1_tbl);
    free(h_sh2_tbl);
    free(h_param_tbl);
    free(h_temper_tbl);
    free(h_single_temper_tbl);
    free(h_mask);
}

/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param[in] d_status kernel I/O data.
 * @param[in] num_data number of data to be generated.
 */
void make_uint32_random(mtgp32_kernel_status_t* d_status,
			int num_data,
			int block_num) {
    uint32_t* d_data;
    uint32_t* h_data;
    hipError_t e;
    float gputime;
    hipEvent_t start;
    hipEvent_t end;

    printf("generating 32-bit unsigned random numbers.\n");
    ccudaMalloc((void**)&d_data, sizeof(uint32_t) * num_data);
    /* cutCreateTimer(&timer); */
    ccudaEventCreate(&start);
    ccudaEventCreate(&end);

    h_data = (uint32_t *) malloc(sizeof(uint32_t) * num_data);
    if (h_data == NULL) {
	printf("failure in allocating host memory for output data.\n");
	exit(1);
    }
    /* cutStartTimer(timer); */
    ccudaEventRecord(start, 0);
    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");
	exit(1);
    }

    /* kernel call */
    mtgp32_uint32_kernel<<< block_num, THREAD_NUM>>>(
	d_status, d_data, num_data / block_num);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	exit(1);
    }
    /* ccutStopTimer(timer); */
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaMemcpy(h_data, d_data,
		sizeof(uint32_t) * num_data, hipMemcpyDeviceToHost);
    /* gputime = cutGetTimerValue(timer); */
    ccudaEventElapsedTime(&gputime, start, end);
    print_uint32_array(h_data, num_data, block_num);
    printf("generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", gputime);
    printf("Samples per second: %E \n", num_data / (gputime * 0.001));

    /* ccutDeleteTimer(timer); */
    ccudaEventDestroy(start);
    ccudaEventDestroy(end);
    //free memories
    free(h_data);
    ccudaFree(d_data);
}

/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param[in] d_status kernel I/O data.
 * @param[in] num_data number of data to be generated.
 */
void make_single_random(mtgp32_kernel_status_t* d_status,
			int num_data,
			int block_num) {
    uint32_t* d_data;
    float* h_data;
    hipError_t e;
    float gputime;
    hipEvent_t start;
    hipEvent_t end;

    printf("generating single precision floating point random numbers.\n");
    ccudaMalloc((void**)&d_data, sizeof(uint32_t) * num_data);
    /* ccutCreateTimer(&timer); */
    ccudaEventCreate(&start);
    ccudaEventCreate(&end);
    h_data = (float *) malloc(sizeof(float) * num_data);
    if (h_data == NULL) {
	printf("failure in allocating host memory for output data.\n");
	exit(1);
    }
    /* ccutStartTimer(timer); */
    ccudaEventRecord(start, 0);
    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");
	exit(1);
    }

    /* kernel call */
    mtgp32_single_kernel<<< block_num, THREAD_NUM >>>(
	d_status, d_data, num_data / block_num);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	exit(1);
    }
    /* ccutStopTimer(timer); */
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaMemcpy(h_data, d_data, sizeof(uint32_t) * num_data,
		hipMemcpyDeviceToHost);
    /* gputime = cutGetTimerValue(timer); */
    ccudaEventElapsedTime(&gputime, start, end);
    print_float_array(h_data, num_data, block_num);
    printf("generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", gputime);
    printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    /* ccutDeleteTimer(timer); */
    ccudaEventDestroy(start);
    ccudaEventDestroy(end);
    //free memories
    free(h_data);
    ccudaFree(d_data);
}

int main(int argc, char** argv)
{
    // LARGE_SIZE is a multiple of 16
    int num_data = 10000000;
    int block_num;
    int num_unit;
    int r;
    int device = 0;
    mtgp32_kernel_status_t *d_status;
    int mb, mp;

    ccudaSetDevice(device);

    if (argc >= 2) {
	errno = 0;
	block_num = strtol(argv[1], NULL, 10);
	if (errno) {
	    printf("%s number_of_block number_of_output\n", argv[0]);
	    return 1;
	}
	if (block_num < 1 || block_num > BLOCK_NUM_MAX) {
	    printf("%s block_num should be between 1 and %d\n",
		   argv[0], BLOCK_NUM_MAX);
	    return 1;
	}
	errno = 0;
	num_data = strtol(argv[2], NULL, 10);
	if (errno) {
	    printf("%s number_of_block number_of_output\n", argv[0]);
	    return 1;
	}
	argc -= 2;
	argv += 2;
    } else {
	printf("%s number_of_block number_of_output\n", argv[0]);
	block_num = get_suitable_block_num(device,
					   &mb,
					   &mp,
					   sizeof(uint32_t),
					   THREAD_NUM,
					   LARGE_SIZE);
	if (block_num <= 0) {
	    printf("can't calculate sutable number of blocks.\n");
	    return 1;
	}
	printf("the suitable number of blocks for device 0 "
	       "will be multiple of %d, or multiple of %d\n", block_num,
	       (mb - 1) * mp);
	return 1;
    }
    num_unit = LARGE_SIZE * block_num;
    ccudaMalloc((void**)&d_status,
			      sizeof(mtgp32_kernel_status_t) * block_num);
    r = num_data % num_unit;
    if (r != 0) {
	num_data = num_data + num_unit - r;
    }
    make_constant(MTGPDC_PARAM_TABLE, block_num);
    make_kernel_data32(d_status, MTGPDC_PARAM_TABLE, block_num);
    make_uint32_random(d_status, num_data, block_num);
    make_single_random(d_status, num_data, block_num);

    ccudaFree(d_status);
}
