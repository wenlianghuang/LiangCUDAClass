#include "hip/hip_runtime.h"
// nvcc main.cu -o sar -lcuda -lm

#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <cassert>
#include <ctime>

#define NICHE_F(h,u,E,mysigma)  (h*exp(-(u-E)*(u-E)/(2*mysigma*mysigma)))
#define EMPTY -1

int main(int argc, char *argv[])
{
    // Seed:
    unsigned int seed = time(NULL);

    // List of variables for run conditions:
    const int num_species = 50;
    const int num_patches = 300;
    const int num_step = 5000;
    const int search_radius = 5;

    // Species characteristics:
    const float d = 0.5; // Species mean dispersal distance (shapes dispersal kernel): 1/d
    const float m = 0.1; // Disturbance probability (disturbance kills individual)
    const float niche_min = 0.1; // Competitive strength of empty cells
    const float emi_from_out = 0.001; // Seed bank

    float *h_matrix_e = (float*)malloc(num_patches * num_patches * sizeof(float));
    int *occupied = (int*)malloc(num_patches * num_patches * sizeof(int));
    int *disturbance_counter = (int*)malloc(num_patches * num_patches * sizeof(int));

    // List of global species characteristic parameters (read in from file or defined in main function):
    float *h = (float*)malloc(num_species * sizeof(float));
    float *u = (float*)malloc(num_species * sizeof(float));
    float *c = (float*)malloc(num_species * sizeof(float));
    float *sigma = (float*)malloc(num_species * sizeof(float));

    // read in species characteristics from file
    char buffer[100];
    sprintf(buffer, "%s_st.txt", argv[1]);
    std::ifstream input_traits(buffer);
    for (int i = 0; i < num_species; ++i)
    {
        input_traits >> sigma[i]; // Niche width
        input_traits >> h[i]; // Performance at niche optimum (height, y at maximum)
        input_traits >> c[i]; // Seed production
        input_traits >> u[i]; // Resource at niche optimum (location, x at maximum)
    }
    input_traits.close();
  
    sprintf(buffer, "%s_species.txt", argv[1]);
    std::ifstream input_species(buffer);
    {
        int i = 0, j = 0;
        while (!input_species.eof())
        {
            input_species >> i;
            input_species >> j;
            input_species >> E[i][j];
            input_species >> occupied[i][j];
            input_species >> disturbance_counter[i][j];
        }
    }
    input_species.close();

    for (int block = 0; block < 1; ++block)
    {
        sprintf(buffer, "%s_dest%d.txt", argv[1], block);
        std::ifstream input_dest(buffer);
        
        for (int i = 0; i < num_patches; ++i)
        {
            for (int j = 0; j < num_patches; ++j)
            {
                disturbance_counter[i][j] = 0;
                //                occupied[i][j] = 0;
            }
        }
        
        // Removed the input_dest as a matrix and replaced by a list (see next loop).
        
        //		for (int i = 0; i < num_patches; ++i)
        //        {
        //            for (int j = 0; j < num_patches; ++j)
        //            {
        //                input_dest >> E[i][j];
        //            }
        //        }
        
        // The input file is now x,y,E with 90000 lines
        
        {
            int i = 0, j = 0;
            while (!input_dest.eof())
            {
                input_dest >> i;
                input_dest >> j;
                input_dest >> E[i][j];
            }
        }
        
        input_dest.close();
        float all_possible_seeds = 0;

        for (int dx = -search_radius; dx <= search_radius; dx++)
        {
            for (int dy = -search_radius; dy <= search_radius; dy++)
            {
                if ((dx != 0) || (dy != 0))
                {
                    all_possible_seeds += exp(-d * sqrt(dx * dx + dy * dy));
                }
            }
        }
        all_possible_seeds += emi_from_out * num_species;

        for (int t = 0; t < num_step; ++t)
        {
            for (int cell = 0; cell < num_patches * num_patches; cell++)
            {
                const int x = (int)(rng.Fixed() * num_patches);
                const int y = (int)(rng.Fixed() * num_patches);

                if(rng.Fixed() < m)
                {
                    occupied[x][y] = EMPTY;
                    ++disturbance_counter[x][y]; // BR
                }

                float Niche_res = niche_min;
                if (occupied[x][y] != EMPTY)
                {
                    int i = occupied[x][y];
                    Niche_res = NICHE_F(h[i], u[i], E[x][y], sigma[i]);
                }

                float *Seed = (float*)calloc(num_species, sizeof(float));

                const int dx_min = (x - search_radius < 0) ? -x : -search_radius; 
                const int dx_max = (x + search_radius >= num_patches) ? num_patches - 1 - x : search_radius;
                const int dy_min = (y - search_radius < 0) ? -y : -search_radius;
                const int dy_max = (y + search_radius >= num_patches) ? num_patches - 1 - y : search_radius; 

                assert(x + dx_min >= 0 && x + dx_max < num_patches);
                assert(y + dy_min >= 0 && y + dy_max < num_patches);

                for (int dx = dx_min; dx <= dx_max; ++dx)
                {
                    for (int dy = dy_min; dy <= dy_max; ++dy)
                    {
                        if (((dx != 0) || (dy != 0)) && (occupied[x + dx][y + dy] != EMPTY))
                        {
                            int i = occupied[x + dx][y + dy];
                            if (NICHE_F(h[i], u[i], E[x][y], sigma[i]) > Niche_res)
                            {
                                Seed[i] += c[i] * (exp(-d * sqrt(dx * dx + dy * dy)));
                            }
                        }
                    }
                }
                float all_seeds = 0.0;
                for (int i = 0; i < num_species; ++i)
                {
                    if (NICHE_F(h[i], u[i], E[x][y], sigma[i]) > Niche_res)
                    {
                        Seed[i] += emi_from_out;
                    }
                    all_seeds += Seed[i];
                }

                bool total_colon = false;
                if (all_seeds / all_possible_seeds > rng.Fixed())
                {
                    total_colon = true;
                }

                float *Prob_recruit = (float*)calloc(num_species, sizeof(float));
                if (total_colon == true)
                {
                    //Prob_recruit[0] = Colon[0]/double(total_colon);
                    Prob_recruit[0] = Seed[0] / all_seeds;
                    for (int i = 1; i < num_species; ++i)
                    {
                        Prob_recruit[i] = Seed[i] / all_seeds + Prob_recruit[i-1];
                    }

                    occupied[x][y] = EMPTY;

                    float randnumb = rng.Fixed();
                    for (int i = 0; i < num_species; i++)
                    {
                        if (randnumb < Prob_recruit[i])
                        {
                            assert(NICHE_F(h[i], u[i], E[x][y], sigma[i]) > niche_min);
                            occupied[x][y] = i;
                            break;
                        }
                    }
                }
                free(Seed);
                free(Prob_recruit);
            } // ends loop over cells
        } // ends loop t

        sprintf(buffer, "%s_out200_%d.txt", argv[1], block);
        std::ofstream out(buffer);
        for (int x = 0; x < num_patches; ++x)
        {
            for (int y = 0; y < num_patches; ++y)
            {
                out << x << " " << y << " " << E[x][y] << " " << occupied[x][y] << " " << disturbance_counter[x][y] << "\n";
            }
        }
        out.close();
    }
    return EXIT_SUCCESS;
}

